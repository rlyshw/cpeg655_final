
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 8
#define logn 3
#define threadsPerBlock 9

__global__ void MatrixMultiply(float *d_A, float *d_B, float *d_C)
{
	
	int ty = threadIdx.y;
	int tx = threadIdx.x;
	
	float Cvalue = 0.0;	
	int k;


	for(k=0; k<N; k++){
		float d_Aelment = d_A[ty*N + k];
		float d_Belment = d_B[k*N + tx];
		Cvalue += d_Aelment * d_Belment;		
	}
	d_C[ty*N + tx]= Cvalue;
}

__global__ void dft(double*x, double*Xre, double*Xim){
	//Credit to Shengfeng Chen from 
	//https://cs.wmich.edu/gupta/teaching/cs5260/5260Sp15web/studentProjects/IMPLEMENTATION%20of%20DFT%20in%20CPU%20and%20GPU%20by%20Shengfeng.pdf
	__shared__ double cache[2*(N-1)];
	int n = threadIdx.x, k=blockIdx.x, cacheIndex = threadIdx.x;
//	Matrix computation for Xim and Xre
	double temp1=0,temp2=0;
	while(n<N && k<N){
		temp1 += x[n] * cos(n*k*(M_PI*2) / N);
		temp2 -= x[n] * sin(n*k*(M_PI*2) / N);
		n+=N; k+=N;
	}

	cache[cacheIndex] = temp1;
	cache[cacheIndex+blockDim.x] = temp2;
	__syncthreads();
	
	int i =blockDim.x/2;
	while(i!=0){
	if(cacheIndex<i){
		cache[cacheIndex]+=cache[cacheIndex+i];
		cache[blockDim.x+cacheIndex]+=cache[blockDim.x+cacheIndex+i];}
	__syncthreads();
	i/=2;
	}
	if(cacheIndex == 0){
		Xre[blockIdx.x] = cache[0];
		Xim[blockIdx.x] = cache[blockDim.x];}
}
__global__ void fft(double*x, double*Xre, double*Xim){
	int tx = threadIdx.x, ty=threadIdx.y;
	double temp = 0;
	const int cache_size = logn*N;
	__shared__ double cache[cache_size];

	if(tx==(logn-1)){
		int ty_rev = __brev(ty)>>(32-logn);
		if ((ty%2)==0) temp = x[ty_rev]+x[ty_rev+N/2];
		else if ((ty%2)!=0) temp = x[ty_rev-N/2]-x[ty_rev];
		//printf("x[%d]:%f tx,ty:%d,%d val:%f ty_rev:%d\n",ty,x[ty],tx,ty,temp,ty_rev);
	}
	else{	
		int n_prime = (int)N/powf(2,tx);
		int offset;
		if(ty<(n_prime/2))
			offset = ty+(n_prime/2);
		else
			offset = ty%(n_prime/2);
		float TF;
		TF = cos(tx*ty*(M_PI*2) / N);
		//printf("tx,ty:%d,%d n_prime:%d offset:%d\n",tx,ty,n_prime,offset);
		__syncthreads();
		if((ty%2)==0) temp = cache[tx+1+logn*ty]+cache[tx+1+logn*offset]*TF;
		if((ty%2)!=0) temp = cache[tx+1+logn*ty]-cache[tx+1+logn*offset]*TF;
		//printf("tx,ty:%d,%d val:%f\n",tx,ty,temp);
	}

	cache[tx+logn*ty] = temp;
	Xre[ty] = cache[logn*ty];
	__syncthreads();
}

int main(){

	int i;
	
	double *d_X, *d_Xre, *d_Xim;	
	double *h_X, *h_Xre, *h_Xim;

	size_t size = N*sizeof(double);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Allocate Device memory
	hipMalloc((void **)&d_X, size);
	hipMalloc((void **)&d_Xre, size);
	hipMalloc((void **)&d_Xim, size);

	//Allocate Host memory
	hipHostMalloc((void **)&h_X, size, hipHostMallocDefault);
	hipHostMalloc((void **)&h_Xre, size, hipHostMallocDefault);	
	hipHostMalloc((void **)&h_Xim, size, hipHostMallocDefault);

	
	//Initialize matrices on the host
	for(i=0;i<N;i++){
		h_X[i]=i;//sin(i);
	}


	//Allocate X to the Device
	hipMemcpy(d_X, h_X, size, hipMemcpyHostToDevice);


	//Invoke kernel
	dim3 blockPerGrid(1);
	dim3 threadPerBlock(logn,N);

	//cudaEventRecord(start);	
	//MatrixMultiply<<<blockPerGrid, threadPerBlock>>>(d_A, d_B, d_C);
	//cudaEventRecord(stop);	

	hipEventRecord(start);	
	fft<<<blockPerGrid, threadPerBlock>>>(d_X, d_Xre, d_Xim);
	hipEventRecord(stop);	


	//Read from device
	hipMemcpy(h_Xre, d_Xre, size, hipMemcpyDeviceToHost);
	hipMemcpy(h_Xim, d_Xim, size, hipMemcpyDeviceToHost);
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("%f\n", milliseconds);
	
	//Calculate the MM result with normal CPU implementation and compare the results with the GPU

	float * test_Xre;
	float * test_Xim;
	test_Xre = (float *)malloc(size);	
	test_Xim = (float *)malloc(size);	
	for (int k=0;k<N;k++){
		test_Xre[k]=0;
		test_Xim[k]=0;
		for(int n=0;n<N;n++){
			test_Xre[k]+=h_X[n]*cos(n*k*M_PI*2 / N);
			test_Xim[k]-=h_X[n]*sin(n*k*M_PI*2 / N);
		}
	}
	int compare_Xre = 0;
	int compare_Xim = 0;
	for(i=0;i<N;i++){
		printf("XRE: test: %f comp: %f\n",test_Xre[i],h_Xre[i]);
		//printf("XIM: test: %f comp: %f\n\n",test_Xim[i],h_Xim[i]);
		if(test_Xre[i]==h_Xre[i]) compare_Xre++;
		if(test_Xim[i]==h_Xim[i]) compare_Xim++;
	}
	if(compare_Xre == N){// && compare_Xim==N){
		printf("Success!\n");
	}else{
		printf("Error!\n");	
	}

	/*=============================Finish Test=================================*/

	free(test_Xre);
	free(test_Xim);
	hipFree(d_X);
	hipFree(d_Xre);
	hipFree(d_Xim);
	hipFree(h_X);
	hipFree(h_Xre);
	hipFree(h_Xim);
	hipDeviceReset();
	return EXIT_SUCCESS;
}
