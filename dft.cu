
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 22 
#define threadsPerBlock 9

__global__ void MatrixMultiply(float *d_A, float *d_B, float *d_C)
{
	
	int ty = threadIdx.y;
	int tx = threadIdx.x;
	
	float Cvalue = 0.0;	
	int k;


	for(k=0; k<N; k++){
		float d_Aelment = d_A[ty*N + k];
		float d_Belment = d_B[k*N + tx];
		Cvalue += d_Aelment * d_Belment;		
	}
	d_C[ty*N + tx]= Cvalue;
}

__global__ void dft(double*x, double*Xre, double*Xim){
	__shared__ double cache[2*N];
	int n = threadIdx.x, k=blockIdx.x, cacheIndex = threadIdx.x;
//	Matrix computation for Xim and Xre
	double temp1=0,temp2=0;
	while(n<N && k<N){
		temp1 += x[n] * cos(n*k*(M_PI*2) / N);
		temp2 -= x[n] * sin(n*k*(M_PI*2) / N);
		n+=N; k+=N;
	}
	cache[cacheIndex] = temp1;
	cache[cacheIndex+blockDim.x] = temp2;
	__syncthreads();
	
	int i =blockDim.x/2;
	while(i!=0){
	if(cacheIndex<i){
		cache[cacheIndex]+=cache[cacheIndex+i];
		cache[blockDim.x+cacheIndex]+=cache[blockDim.x+cacheIndex+i];}
	__syncthreads();
	i/=2;
	}
	if(cacheIndex == 0){
		Xre[blockIdx.x] = cache[0];
		Xim[blockIdx.x] = cache[blockDim.x];}
}

int main(){

	int i,j;
	
	double *d_X, *d_Xre, *d_Xim;	
	double *h_X, *h_Xre, *h_Xim;

	size_t size = N*sizeof(double);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Allocate Device memory
	hipMalloc((void **)&d_X, size);
	hipMalloc((void **)&d_Xre, size);
	hipMalloc((void **)&d_Xim, size);

	//Allocate Host memory
	hipHostMalloc((void **)&h_X, size, hipHostMallocDefault);
	hipHostMalloc((void **)&h_Xre, size, hipHostMallocDefault);	
	hipHostMalloc((void **)&h_Xim, size, hipHostMallocDefault);

	
	//Initialize matrices on the host
	for(i=0;i<N;i++){
	    for(j=0;j<N;j++){
		h_X[i*N+j]=i;
	    }
	}


	//Allocate X to the Device
	hipMemcpy(d_X, h_X, size, hipMemcpyHostToDevice);
	//cudaMemcpy(d_B, h_B, size, cudaMemcpyHostToDevice);


	//Invoke kernel
	dim3 blockPerGrid(N,1);
	dim3 threadPerBlock(N,1);

	//cudaEventRecord(start);	
	//MatrixMultiply<<<blockPerGrid, threadPerBlock>>>(d_A, d_B, d_C);
	//cudaEventRecord(stop);	

	hipEventRecord(start);	
	dft<<<blockPerGrid, threadPerBlock>>>(d_X, d_Xre, d_Xim);
	hipEventRecord(stop);	


	//Read from device
	hipMemcpy(h_Xre, d_Xre, size, hipMemcpyDeviceToHost);
	hipMemcpy(h_Xim, d_Xim, size, hipMemcpyDeviceToHost);
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("%f\n", milliseconds);
	
	//Calculate the MM result with normal CPU implementation and compare the results with the GPU

	float * test_Xre;
	float * test_Xim;
	test_Xre = (float *)malloc(size);	
	test_Xim = (float *)malloc(size);	
	for (int k=0;k<N;k++){
		test_Xre[k]=0;
		test_Xim[k]=0;
		for(int n=0;n<N;n++){
			test_Xre[k]+=h_X[n]*cos(n*k*M_PI*2 / N);
			test_Xim[k]+=h_X[n]*cos(n*k*M_PI*2 / N);
		}
	}
	int compare_Xre = 0;
	int compare_Xim = 0;
	for(i=0;i<N;i++){
		if(test_Xre[i]==h_Xre[i]) compare_Xre++;
		if(test_Xim[i]==h_Xim[i]) compare_Xim++;
	}
	if(compare_Xre == N && compare_Xim==N){
		printf("Success!\n");
	}else{
		printf("Error!\n");	
	}

	/*=============================Finish Test=================================*/

	free(test_Xre);
	free(test_Xim);
	hipFree(d_X);
	hipFree(d_Xre);
	hipFree(d_Xim);
	hipFree(h_X);
	hipFree(h_Xre);
	hipFree(h_Xim);
	hipDeviceReset();
	return EXIT_SUCCESS;
}